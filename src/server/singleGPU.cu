#include "hip/hip_runtime.h"
/*
  For the Time being just use a single dimensional array.
*/
#include <stdlib.h>
#include <stdio.h>

#include <cutil_inline.h>
#include "gpu_kernels.cuh"

#define MAX_ITERATIONS 400

/* Used by computeDist.c */
float distanceEuclidean(float* refPoint, float* testPoint, int numDimensions){

  float sum = 0.0;
  int dim;

  for(dim = 0; dim < numDimensions; dim++)
    sum += (refPoint[dim] - testPoint[dim]) * (refPoint[dim] - testPoint[dim]);

  return sqrt(sum);
}

/* Used by computeError.c */
int errorAbsDifference(int newVal, int oldVal){

  return abs(newVal - oldVal);
}

int errorCheckEquality(int newVal, int oldVal){
  return newVal == oldVal ? 0 : 1;
}

extern "C" void singleGPU(float* objects, int numDataPoints, 
	       int numDimensions, float* clusters,
	       int numClusters, int *membership,
			  int *newClusterSize,
	       float* newClusters, float *error)
{
  float* d_dataPoints;
  float* d_clusterCentres;
  int* d_membership;
  int* d_clusterHistogram;
  int* h_clusterHistogram = (int*)malloc(sizeof(int)*numClusters);
  int i;

  //int* h_clusterHistogram = (int*)malloc(sizeof(int)*numClusters);

  /* Initialize the GPU device */
  CUT_DEVICE_INIT(1, "pvfs2-server");
  
  /* Print out Kmeans parameters */
  fprintf(stderr, "[INFO] Records: %d, Dimensions: %d, Clusters: %d\n", numDataPoints, numDimensions, numClusters);

  for(i=0; i<numDimensions; i++)
    fprintf(stderr, "object[%d]=%f\n", i, objects[i]);

  for(i=0; i<numDimensions; i++)
    fprintf(stderr, "clusters[%d]=%f\n", i, clusters[i]);

  /* Allocate memory on the device and copy the data */
  fprintf(stderr, "Memory Initialization ... [START]\n");
  
  hipMalloc((void**) &d_dataPoints, sizeof(float)*numDataPoints*numDimensions);
  hipMalloc((void**) &d_clusterCentres, sizeof(float)*numClusters*numDimensions);
  hipMalloc((void**) &d_membership, sizeof(int)*numDataPoints);
  hipMalloc((void**) &d_clusterHistogram, sizeof(int)*numClusters);

  /* Memory Initialization */
  hipMemcpy(d_dataPoints, objects, sizeof(float)*numDataPoints*numDimensions, hipMemcpyHostToDevice);
  hipMemcpy(d_clusterCentres, clusters, sizeof(float)*numClusters*numDimensions, hipMemcpyHostToDevice);
  hipMemcpy(d_membership, membership, sizeof(int)*numDataPoints, hipMemcpyHostToDevice);

  fprintf(stderr, "Memory Initialization ... [DONE]\n");

  /* Compute the distance */
  gpuComputeDistance(d_dataPoints, numDataPoints, numDimensions, d_clusterCentres, numClusters, d_membership, distanceEuclidean);
  fprintf(stderr, "gpuComputeDistance ... [DONE]\n");
  hipMemcpy(membership, d_membership, sizeof(int)*numDataPoints, hipMemcpyDeviceToHost);
  for(i=0; i<22; i++)
    fprintf(stderr, "d_membership[%d]=%d\n", i, membership[i]);

  /* Compute the Histogram */
  gpuCountClusterPoints(d_membership, numDataPoints, numClusters, d_clusterHistogram);
  hipMemcpy(h_clusterHistogram, d_clusterHistogram, sizeof(int)*numClusters, hipMemcpyDeviceToHost);
  for(i=0; i<numClusters; i++)
    fprintf(stderr, "d_clusterHistogram[%d]=%d\n", i, h_clusterHistogram[i]);
  fprintf(stderr, "gpuCountClusterPoints ... [DONE]\n");

  /* Update the Cluster Centres, using the d_clusterHistogram */
  //[MODIFY] I am for the time being updating the update cluster to give out the average value rather than summation
  gpuUpdateClusterCentres(d_membership, d_dataPoints, numDataPoints, numDimensions, numClusters, d_clusterHistogram, d_clusterCentres);
  fprintf(stderr, "gpuUpdateClusterCentres ... [DONE]\n");

  /* Compute the Error */
  //gpuComputeError(d_membership, d_oldMembership, numDataPoints, error);
  //fprintf(stderr, "gpuComputeError ... [DONE]\n");

  /* Copy back the result to the host */
  hipMemcpy(newClusters, d_clusterCentres, sizeof(float)*numClusters*numDimensions, hipMemcpyDeviceToHost);
  hipMemcpy(newClusterSize, d_clusterHistogram, sizeof(int)*numClusters, hipMemcpyDeviceToHost);
  for(i=0; i<numDimensions; i++)
    fprintf(stderr, "newClusters[%d]=%f\n", i, newClusters[i]);
  hipMemcpy(membership, d_membership, sizeof(int)*numDataPoints, hipMemcpyDeviceToHost);

  /* Free the device memory */
  hipFree(d_dataPoints);
  hipFree(d_clusterCentres);
  hipFree(d_membership);
  hipFree(d_clusterHistogram);

  hipDeviceReset();

  //fprintf(stderr, "[RESULT] Total Execution time = %f ms\n", cutGetTimerValue(timer));

  /* End Program Execution */
}
